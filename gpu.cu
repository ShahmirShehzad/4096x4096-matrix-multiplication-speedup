#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#include <math.h>

#define DIMENSION 4096
#define BLOCK_SIZE 16


// CUDA kernel for matrix multiplication (32-bit integer)
__global__ void matrixMul32(int *matrix1, int *matrix2, int *matrix3) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;

    if (row < DIMENSION && col < DIMENSION) {
        for (int k = 0; k < DIMENSION; ++k) {
            sum += matrix1[row * DIMENSION + k] * matrix2[k * DIMENSION + col];
        }
        matrix3[row * DIMENSION + col] = sum;
    }
}

// CUDA kernel for matrix multiplication (64-bit integer)
__global__ void matrixMul64(long long int *matrix1, long long int *matrix2, long long int *matrix3) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
int col = blockIdx.x * blockDim.x + threadIdx.x;
    long long int sum = 0;

    if (row < DIMENSION && col < DIMENSION) {
        for (int k = 0; k < DIMENSION; ++k) {
            sum += matrix1[row * DIMENSION + k] * matrix2[k * DIMENSION + col];
        }
        matrix3[row * DIMENSION + col] = sum;
    }
}

// CUDA kernel for matrix multiplication (32-bit float)
__global__ void matrixMulFloat(float *matrix1, float *matrix2, float *matrix3) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    float sum = 0.0f;

    if (row < DIMENSION && col < DIMENSION) {
        for (int k = 0; k < DIMENSION; ++k) {
            sum += matrix1[row * DIMENSION + k] * matrix2[k * DIMENSION + col];
        }
        matrix3[row * DIMENSION + col] = sum;
    }
}

// CUDA kernel for matrix multiplication (64-bit double)
__global__ void matrixMulDouble(double *matrix1, double *matrix2, double *matrix3) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    double sum = 0.0;

    if (row < DIMENSION && col < DIMENSION) {
        for (int k = 0; k < DIMENSION; ++k) {
            sum += matrix1[row * DIMENSION + k] * matrix2[k * DIMENSION + col];
        }
        matrix3[row * DIMENSION + col] = sum;
    }
}

// Function to perform matrix multiplication on the GPU
void matrixMultiplyGPU32int(int *matrix1, int *matrix2, int *matrix3) {
    int *d_matrix1, *d_matrix2, *d_matrix3;

    // Allocate memory on the device
    hipMalloc((int **)&d_matrix1, DIMENSION * DIMENSION * sizeof(int));
    hipMalloc((int **)&d_matrix2, DIMENSION * DIMENSION * sizeof(int));
    hipMalloc((int **)&d_matrix3, DIMENSION * DIMENSION * sizeof(int));

    // Copy input matrices from host to device
    hipMemcpy(d_matrix1, matrix1, DIMENSION * DIMENSION * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_matrix2, matrix2, DIMENSION * DIMENSION * sizeof(int), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((DIMENSION + BLOCK_SIZE - 1) / BLOCK_SIZE, (DIMENSION + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Launch kernel based on data type
    matrixMul32<<<gridSize, blockSize>>>(d_matrix1, d_matrix2, d_matrix3);

    // Copy the result matrix from device to host
    hipMemcpy(matrix3, d_matrix3, DIMENSION * DIMENSION * sizeof(int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_matrix1);
    hipFree(d_matrix2);
    hipFree(d_matrix3);
}

void matrixMultiplyGPU64int(long long int *matrix1, long long int *matrix2, long long int *matrix3) {
    long long int *d_matrix1, *d_matrix2, *d_matrix3;

    // Allocate memory on the device
    hipMalloc((void **)&d_matrix1, DIMENSION * DIMENSION * sizeof(long long int));
    hipMalloc((void **)&d_matrix2, DIMENSION * DIMENSION * sizeof(long long int));
    hipMalloc((void **)&d_matrix3, DIMENSION * DIMENSION * sizeof(long long int));

    // Copy input matrices from host to device
    hipMemcpy(d_matrix1, matrix1, DIMENSION * DIMENSION * sizeof(long long int), hipMemcpyHostToDevice);
    hipMemcpy(d_matrix2, matrix2, DIMENSION * DIMENSION * sizeof(long long int), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((DIMENSION + BLOCK_SIZE - 1) / BLOCK_SIZE, (DIMENSION + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Launch kernel based on data type
    matrixMul64<<<gridSize, blockSize>>>(d_matrix1, d_matrix2, d_matrix3);

    // Copy the result matrix from device to host
    hipMemcpy(matrix3, d_matrix3, DIMENSION * DIMENSION * sizeof(long long int), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_matrix1);
    hipFree(d_matrix2);
    hipFree(d_matrix3);
}

void matrixMultiplyGPUfloat(float *matrix1, float *matrix2, float *matrix3) {
    float *d_matrix1, *d_matrix2, *d_matrix3;

    // Allocate memory on the device
    hipMalloc((void **)&d_matrix1, DIMENSION * DIMENSION * sizeof(float));
    hipMalloc((void **)&d_matrix2, DIMENSION * DIMENSION * sizeof(float));
    hipMalloc((void **)&d_matrix3, DIMENSION * DIMENSION * sizeof(float));

    // Copy input matrices from host to device
    hipMemcpy(d_matrix1, matrix1, DIMENSION * DIMENSION * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_matrix2, matrix2, DIMENSION * DIMENSION * sizeof(float), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((DIMENSION + BLOCK_SIZE - 1) / BLOCK_SIZE, (DIMENSION + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Launch kernel based on data type
    matrixMulFloat<<<gridSize, blockSize>>>(d_matrix1, d_matrix2, d_matrix3);

    // Copy the result matrix from device to host
    hipMemcpy(matrix3, d_matrix3, DIMENSION * DIMENSION * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_matrix1);
    hipFree(d_matrix2);
    hipFree(d_matrix3);
}

void matrixMultiplyGPUdouble(double *matrix1, double *matrix2, double *matrix3) {
    double *d_matrix1, *d_matrix2, *d_matrix3;

    // Allocate memory on the device
    hipMalloc((void **)&d_matrix1, DIMENSION * DIMENSION * sizeof(double));
    hipMalloc((void **)&d_matrix2, DIMENSION * DIMENSION * sizeof(double));
    hipMalloc((void **)&d_matrix3, DIMENSION * DIMENSION * sizeof(double));

    // Copy input matrices from host to device
    hipMemcpy(d_matrix1, matrix1, DIMENSION * DIMENSION * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_matrix2, matrix2, DIMENSION * DIMENSION * sizeof(double), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((DIMENSION + BLOCK_SIZE - 1) / BLOCK_SIZE, (DIMENSION + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Launch kernel based on data type
    matrixMulDouble<<<gridSize, blockSize>>>(d_matrix1, d_matrix2, d_matrix3);

    // Copy the result matrix from device to host
    hipMemcpy(matrix3, d_matrix3, DIMENSION * DIMENSION * sizeof(double), hipMemcpyDeviceToHost);

    // Free device memory
    hipFree(d_matrix1);
    hipFree(d_matrix2);
    hipFree(d_matrix3);
}

void Run_32BitInt(char* input_file, char* output_file)
{
    	FILE* fptr = fopen(input_file, "r");
    	
    	if (fptr == NULL)
    	{
    		printf("File could not be opened\n");
    		exit(0);
    	}
    	
    	int programtype = 0;
    	int datatype = 0;
    	int dimension1 = 0, dimension2 = 0;
    	char garbage;
    	
    	fscanf(fptr, "%d", &programtype);
    	fscanf(fptr, "%d", &datatype);
    	fscanf(fptr, "%d%c%d", &dimension1, &garbage, &dimension2);
    	
    	if(dimension1 != 4096 || dimension2 != 4096)
    	{
    		printf("Invalid Dimensions given for matrix 1\n");
    		printf("DImension 1: %d", dimension1);
    		printf("DImension 2: %d", dimension2);
    		exit(0);
    	}
    	
    	int i = 0, j = 0;
    	
    	
    	
    	static int matrix1[4096][4096];
    	static int matrix2[4096][4096];
    	static int matrix3[4096][4096];
    	
    	for (i = 0; i < 4096; i++)
    	{
    		for (j = 0; j < 4096; j++)
    		{
    			fscanf(fptr, "%d", &matrix1[i][j]);
    		}
    	}
    	
    	fscanf(fptr, "%d%c%d", &dimension1, &garbage, &dimension2);
    	
    	if(dimension1 != 4096 || dimension2 != 4096)
    	{
    		printf("Invalid Dimensions given for matrix 1\n");
    		printf("DImension 1: %d\n", dimension1);
    		printf("DImension 2: %d\n", dimension2);
    		exit(0);
    	}
    	
    	for (i = 0; i < 4096; i++)
    	{
    		for (j = 0; j < 4096; j++)
    		{
    			fscanf(fptr, "%d", &matrix2[i][j]);
    		}
    	}
    	
    	fclose(fptr);
    	
    	clock_t seconds_start;
    	clock_t seconds_finish;
    	
    	int k = 0;
    	
    	int task_size = 4096;
    	int task = 0;
    	
    	seconds_start = time(NULL);

    	printf("%d %d \n",matrix1[0][0],matrix1[3][0]);
    	matrixMultiplyGPU32int((int *)matrix1, (int *)matrix2, (int *)matrix3);
    	
    	seconds_finish = time(NULL);
    	
    	double cputime = ((double) (seconds_finish - seconds_start)) / CLOCKS_PER_SEC * 1000;
    	printf("Time taken for calculations = %f\n", cputime);
    	
    	//printf("%d %d %d", matrix3[1044][1044], matrix3[2067][2067], matrix3[4004][0] );
    	
    	fptr = fopen(output_file, "w");
    	
    	fprintf(fptr, "1\n4096X4096\n");
    	
    	for (i = 0; i < 4096; i++)
    	{
    		for (j = 0; j < 4096; j++)
    		{
    			fprintf(fptr, "%d ", matrix3[i][j]);
    		}
    		fprintf(fptr, " \n");
    	}
    	
    	fclose(fptr);
    	
}
void Run_64BitInt(char* input_file, char* output_file)
{
	FILE* fptr = fopen(input_file, "r");
    	
    	if (fptr == NULL)
    	{
    		printf("File could not be opened\n");
    		exit(0);
    	}
    	
    	int programtype = 0;
    	int datatype = 0;
    	int dimension1 = 0, dimension2 = 0;
    	char garbage;
    	
    	fscanf(fptr, "%d", &programtype);
    	fscanf(fptr, "%d", &datatype);
    	fscanf(fptr, "%d%c%d", &dimension1, &garbage, &dimension2);
    	
    	if(dimension1 != 4096 || dimension2 != 4096 )
    	{
    		printf("Invalid Dimensions given for matrix 1\n");
    		printf("DImension 1: %d", dimension1);
    		printf("DImension 2: %d", dimension2);
    		exit(0);
    	}
    	
    	int i = 0, j = 0;
    	
    	static long long int matrix1[4096][4096];
    	static long long int matrix2[4096][4096];
    	static long long int matrix3[4096][4096];
    	
    	for (i = 0; i < 4096; i++)
    	{
    		for (j = 0; j < 4096; j++)
    		{
    			fscanf(fptr, "%lli", &matrix1[i][j]);
    		}
    	}
    	
    	fscanf(fptr, "%d%c%d", &dimension1, &garbage, &dimension2);
    	
    	if(dimension1 != 4096 || dimension2 != 4096)
    	{
    		printf("Invalid Dimensions given for matrix 1\n");
    		printf("DImension 1: %d\n", dimension1);
    		printf("DImension 2: %d\n", dimension2);
    		exit(0);
    	}
    	
    	for (i = 0; i < 4096; i++)
    	{
    		for (j = 0; j < 4096; j++)
    		{
    			fscanf(fptr, "%lli", &matrix2[i][j]);
    		}
    	}
    	
    	fclose(fptr);
    	
    	clock_t seconds_start;
    	clock_t seconds_finish;
    	
    	int k = 0;
    	
    	int task_size = 4096;
    	int task = 0;
    	
    	seconds_start = time(NULL);

    	
    	matrixMultiplyGPU64int((long long int *)matrix1, (long long int *)matrix2, (long long int *)matrix3);
    	
    	seconds_finish = time(NULL);
    	
    	double cputime = ((double) (seconds_finish - seconds_start)) / CLOCKS_PER_SEC * 1000;
    	printf("Time taken for calculations = %f\n", cputime);
    	
    	fptr = fopen(output_file, "w");
    	
    	fprintf(fptr, "3\n4096X4096\n");
    	
    	for (i = 0; i < 4096; i++)
    	{
    		for (j = 0; j < 4096; j++)
    		{
    			fprintf(fptr, "%lli ", matrix3[i][j]);
    		}
    		fprintf(fptr, " \n");
    	}
    	
    	fclose(fptr);
}
void Run_32BitFloat(char* input_file, char* output_file)
{
	FILE* fptr = fopen(input_file, "r");
    	
    	if (fptr == NULL)
    	{
    		printf("File could not be opened\n");
    		exit(0);
    	}
    	
    	int programtype = 0;
    	int datatype = 0;
    	int dimension1 = 0, dimension2 = 0;
    	char garbage;
    	
    	fscanf(fptr, "%d", &programtype);
    	fscanf(fptr, "%d", &datatype);
    	fscanf(fptr, "%d%c%d", &dimension1, &garbage, &dimension2);
    	
    	if(dimension1 != 4096 || dimension2 != 4096)
    	{
    		printf("Invalid Dimensions given for matrix 1\n");
    		printf("DImension 1: %d", dimension1);
    		printf("DImension 2: %d", dimension2);
    		exit(0);
    	}
    	
    	int i = 0, j = 0;
    	
    	static float matrix1[4096][4096];
    	static float matrix2[4096][4096];
    	static float matrix3[4096][4096];
    	
    	for (i = 0; i < 4096; i++)
    	{
    		for (j = 0; j < 4096; j++)
    		{
    			fscanf(fptr, "%g", &matrix1[i][j]);
    		}
    	}
    	
    	fscanf(fptr, "%d%c%d", &dimension1, &garbage, &dimension2);
    	
    	if(dimension1 != 4096 || dimension2 != 4096)
    	{
    		printf("Invalid Dimensions given for matrix 1\n");
    		printf("DImension 1: %d\n", dimension1);
    		printf("DImension 2: %d\n", dimension2);
    		exit(0);
    	}
    	
    	for (i = 0; i < 4096; i++)
    	{
    		for (j = 0; j < 4096; j++)
    		{
    			fscanf(fptr, "%g", &matrix2[i][j]);
    		}
    	}
    	
    	fclose(fptr);
    	
    	clock_t seconds_start;
    	clock_t seconds_finish;
    	
    	int k = 0;
    	
    	int task_size = 4096;
    	int task = 0;
    	
    	seconds_start = time(NULL);

    	
    	matrixMultiplyGPUfloat((float *)matrix1, (float *)matrix2, (float *)matrix3);
    	
    	
    	seconds_finish = time(NULL);
    	
    	double cputime = ((double) (seconds_finish - seconds_start)) / CLOCKS_PER_SEC * 1000;
    	printf("Time taken for calculations = %f\n", cputime);
    	
    	fptr = fopen(output_file, "w");
    	
    	fprintf(fptr, "2\n4096X4096\n");
    	
    	for (i = 0; i < 4096; i++)
    	{
    		for (j = 0; j < 4096; j++)
    		{
    			fprintf(fptr, "%g ", matrix3[i][j]);
    		}
    		fprintf(fptr, " \n");
    	}
    	
    	fclose(fptr);
}
void Run_64BitDouble(char* input_file, char* output_file)
{
	FILE* fptr = fopen(input_file, "r");
    	
    	if (fptr == NULL)
    	{
    		printf("File could not be opened\n");
    		exit(0);
    	}
    	
    	int programtype = 0;
    	int datatype = 0;
    	int dimension1 = 0, dimension2 = 0;
    	char garbage;
    	
    	fscanf(fptr, "%d", &programtype);
    	fscanf(fptr, "%d", &datatype);
    	fscanf(fptr, "%d%c%d", &dimension1, &garbage, &dimension2);
    	
    	if(dimension1 != 4096 || dimension2 != 4096)
    	{
    		printf("Invalid Dimensions given for matrix 1\n");
    		printf("DImension 1: %d", dimension1);
    		printf("DImension 2: %d", dimension2);
    		exit(0);
    	}
    	
    	int i = 0, j = 0;
    	
    	static double matrix1[4096][4096];
    	static double matrix2[4096][4096];
    	static double matrix3[4096][4096];
    	
    	for (i = 0; i < 4096; i++)
    	{
    		for (j = 0; j < 4096; j++)
    		{
    			fscanf(fptr, "%lf", &matrix1[i][j]);
    		}
    	}
    	
    	fscanf(fptr, "%d%c%d", &dimension1, &garbage, &dimension2);
    	
    	if(dimension1 != 4096 || dimension2 != 4096)
    	{
    		printf("Invalid Dimensions given for matrix 1\n");
    		printf("DImension 1: %d\n", dimension1);
    		printf("DImension 2: %d\n", dimension2);
    		exit(0);
    	}
    	
    	for (i = 0; i < 4096; i++)
    	{
    		for (j = 0; j < 4096; j++)
    		{
    			fscanf(fptr, "%lf", &matrix2[i][j]);
    		}
    	}
    	
    	fclose(fptr);
    	
    	clock_t seconds_start;
    	clock_t seconds_finish;
    	
    	int k = 0;
    	
    	int task_size = 4096;
    	int task = 0;
    	
    	seconds_start = time(NULL);

    	
    	matrixMultiplyGPUdouble((double *)matrix1, (double *)matrix2, (double *)matrix3);
    	
    	
    	seconds_finish = time(NULL);
    	
    	double cputime = ((double) (seconds_finish - seconds_start)) / CLOCKS_PER_SEC * 1000;
    	printf("Time taken for calculations = %f\n", cputime);
    	
    	fptr = fopen(output_file, "w");
    	
    	fprintf(fptr, "4\n4096X4096\n");
    	
    	for (i = 0; i < 4096; i++)
    	{
    		for (j = 0; j < 4096; j++)
    		{
    			fprintf(fptr, "%g ", matrix3[i][j]);
    		}
    		fprintf(fptr, " \n");
    	}
    	
    	fclose(fptr);
}

int main(int argc, char** argv)
{
	if(argc != 3)
    	{
        	printf("2 command line arguments are expected to run the code. Exiting the code.\n");
        	return -1;
    	}
    	
    	char* input_file = argv[1];
    	char* output_file = argv[2];
    	
    	FILE* fptr = fopen(input_file, "r");
    	
    	if (fptr == NULL)
    	{
    		printf("File could not be opened\n");
    		exit(0);
    	}
    	
    	int programtype = 0;
    	int datatype = 0;
    	
    	fscanf(fptr, "%i", &programtype);
    	fscanf(fptr, "%i", &datatype);
    	
    	printf("ProgramType: %i\n", programtype);
    	printf("Data Type: %i\n", datatype);
    	
    	if(datatype == 1)
    	{
    		Run_32BitInt(input_file, output_file);
    	}
    	else if (datatype == 2)
    	{
    		Run_32BitFloat(input_file, output_file);
	}
	else if (datatype == 3)
    	{
    		Run_64BitInt(input_file, output_file);
	}
	else if (datatype == 4)
    	{
    		Run_64BitDouble(input_file, output_file);
	}
	else
	{
		printf("Invalid data type OpCode, Halting execution\n");
		exit(0);
	}
    	
    	
    	
    	
    	    	
    	return 0;
    	
}
